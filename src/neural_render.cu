#include "hip/hip_runtime.h"
#include <float.h>
#include <math.h>

#define EIGEN_USE_GPU
#include "neural_render.h"

// is_on_left_side
template <typename T>
XINLINE bool is_on_left_side(const T *p, const T *a, const T *b) {
  T data[9] = {a[0], a[1], 1, b[0], b[1], 1, p[0], p[1], 1};
  T tmp1 = data[0 * 3 + 0] * (data[1 * 3 + 1] * data[2 * 3 + 2] -
                              data[1 * 3 + 2] * data[2 * 3 + 1]);
  T tmp2 = data[0 * 3 + 1] * (data[1 * 3 + 0] * data[2 * 3 + 2] -
                              data[1 * 3 + 2] * data[2 * 3 + 0]);
  T tmp3 = data[0 * 3 + 2] * (data[1 * 3 + 0] * data[2 * 3 + 1] -
                              data[1 * 3 + 1] * data[2 * 3 + 0]);
  return tmp1 - tmp2 + tmp3 >= 0;
}

// is_in_triangle
template <typename T>
XINLINE bool is_in_triangle(const T *p, const T *a, const T *b, const T *c) {
  bool lab = is_on_left_side(p, a, b);
  bool lbc = is_on_left_side(p, b, c);
  bool lca = is_on_left_side(p, c, a);
  return lab == lbc && lbc == lca;
}

template <typename T>
XINLINE bool between(T value, int lowerBound, int upperBound) {
  return (value >= lowerBound && value <= upperBound);
}

// bc[0]*(b-a) + bc[1]*(c-a) = p-a
// p = (1-bc[0]-bc[1])*a + bc[0]*b + bc[1]*c
template <typename T>
XINLINE void get_barycentric_coord(const T *p, const T *a, const T *b,
                                   const T *c, T *bc) {
  // clang-format off
    /* #bc[0]:
       bx cy - by cx - bx py + by px + cx py - cy px
       ---------------------------------------------
       ax by - ay bx - ax cy + ay cx + bx cy - by cx

       #bc[1]:
       ax cy - ay cx - ax py + ay px + cx py - cy px
     - ---------------------------------------------
       ax by - ay bx - ax cy + ay cx + bx cy - by cx

       #bc[2]:
       ax by - ay bx - ax py + ay px + bx py - by px
       ---------------------------------------------
       ax by - ay bx - ax cy + ay cx + bx cy - by cx
     */
  // clang-format on
  T ax = a[0], ay = a[1];
  T bx = b[0], by = b[1];
  T cx = c[0], cy = c[1];
  T px = p[0], py = p[1];
  T s = ax * by - ay * bx - ax * cy + ay * cx + bx * cy - by * cx;
  if (abs(s) < 1e-6) {
    bc[0] = bc[1] = bc[2] = 1.0 / 3.0;
  } else {
    bc[0] = (bx * cy - by * cx - bx * py + by * px + cx * py - cy * px) / s;
    bc[1] = (ax * cy - ay * cx - ax * py + ay * px + cx * py - cy * px) / (-s);
    bc[2] = (ax * by - ay * bx - ax * py + ay * px + bx * py - by * px) / s;
  }
}

template <typename T>
XINLINE void add_barycentric_coord_grad(const T *p, const T *a, const T *b,
                                        const T *c, const T *grad_bc, T *grad_a,
                                        T *grad_b, T *grad_c) {
  T ax = a[0], ay = a[1];
  T bx = b[0], by = b[1];
  T cx = c[0], cy = c[1];
  T px = p[0], py = p[1];
  T s = ax * by - ay * bx - ax * cy + ay * cx + bx * cy - by * cx;
  if (abs(s) < 1e-6) {
    return;
  }

  T grad_bc0_ax = -(by - cy) * 1.0 / squared(s) *
                  (bx * cy - by * cx - bx * py + by * px + cx * py - cy * px);
  T grad_bc0_ay = (bx - cx) * 1.0 / squared(s) *
                  (bx * cy - by * cx - bx * py + by * px + cx * py - cy * px);
  T grad_bc0_bx = (by - cy) * 1.0 / squared(s) *
                  (ax * cy - ay * cx - ax * py + ay * px + cx * py - cy * px);
  T grad_bc0_by = -(bx - cx) * 1.0 / squared(s) *
                  (ax * cy - ay * cx - ax * py + ay * px + cx * py - cy * px);
  T grad_bc0_cx = -(by - cy) * 1.0 / squared(s) *
                  (ax * by - ay * bx - ax * py + ay * px + bx * py - by * px);
  T grad_bc0_cy = (bx - cx) * 1.0 / squared(s) *
                  (ax * by - ay * bx - ax * py + ay * px + bx * py - by * px);

  T grad_bc1_ax = (ay - cy) * 1.0 / squared(s) *
                  (bx * cy - by * cx - bx * py + by * px + cx * py - cy * px);
  T grad_bc1_ay = -(ax - cx) * 1.0 / squared(s) *
                  (bx * cy - by * cx - bx * py + by * px + cx * py - cy * px);
  T grad_bc1_bx = -(ay - cy) * 1.0 / squared(s) *
                  (ax * cy - ay * cx - ax * py + ay * px + cx * py - cy * px);
  T grad_bc1_by = (ax - cx) * 1.0 / squared(s) *
                  (ax * cy - ay * cx - ax * py + ay * px + cx * py - cy * px);
  T grad_bc1_cx = (ay - cy) * 1.0 / squared(s) *
                  (ax * by - ay * bx - ax * py + ay * px + bx * py - by * px);
  T grad_bc1_cy = -(ax - cx) * 1.0 / squared(s) *
                  (ax * by - ay * bx - ax * py + ay * px + bx * py - by * px);

  T grad_bc2_ax = -(ay - by) * 1.0 / squared(s) *
                  (bx * cy - by * cx - bx * py + by * px + cx * py - cy * px);
  T grad_bc2_ay = (ax - bx) * 1.0 / squared(s) *
                  (bx * cy - by * cx - bx * py + by * px + cx * py - cy * px);
  T grad_bc2_bx = (ay - by) * 1.0 / squared(s) *
                  (ax * cy - ay * cx - ax * py + ay * px + cx * py - cy * px);
  T grad_bc2_by = -(ax - bx) * 1.0 / squared(s) *
                  (ax * cy - ay * cx - ax * py + ay * px + cx * py - cy * px);
  T grad_bc2_cx = -(ay - by) * 1.0 / squared(s) *
                  (ax * by - ay * bx - ax * py + ay * px + bx * py - by * px);
  T grad_bc2_cy = (ax - bx) * 1.0 / squared(s) *
                  (ax * by - ay * bx - ax * py + ay * px + bx * py - by * px);

  T grad_ax = grad_bc[0] * grad_bc0_ax + grad_bc[1] * grad_bc1_ax +
              grad_bc[2] * grad_bc2_ax;
  T grad_ay = grad_bc[0] * grad_bc0_ay + grad_bc[1] * grad_bc1_ay +
              grad_bc[2] * grad_bc2_ay;
  T grad_bx = grad_bc[0] * grad_bc0_bx + grad_bc[1] * grad_bc1_bx +
              grad_bc[2] * grad_bc2_bx;
  T grad_by = grad_bc[0] * grad_bc0_by + grad_bc[1] * grad_bc1_by +
              grad_bc[2] * grad_bc2_by;
  T grad_cx = grad_bc[0] * grad_bc0_cx + grad_bc[1] * grad_bc1_cx +
              grad_bc[2] * grad_bc2_cx;
  T grad_cy = grad_bc[0] * grad_bc0_cy + grad_bc[1] * grad_bc1_cy +
              grad_bc[2] * grad_bc2_cy;

  grad_a[0] += grad_ax;
  grad_a[1] += grad_ay;
  grad_b[0] += grad_bx;
  grad_b[1] += grad_by;
  grad_c[0] += grad_cx;
  grad_c[1] += grad_cy;
}

XGLOBAL void
rasterize_kernel(int batch_size, //
                 int npixeliter_each_block, int nfaceiter_each_block,
                 int npoints, int nfaces,
                 const float *pts_data,     // batch_size x npoints x 3
                 const int32_t *faces_data, // batch_size x nfaces x 3
                 const float *uvs_data,     // batch_size x nfaces x 3 x 2
                 int H, int W,
                 float *out_uvgrid_data, // batch_size x H x W x 2
                 float *out_z_data,      // batch_size x H x W
                 int32_t *out_fids_data, // batch_size x H x W
                 float *out_bc_data      // batch_size x H x W x 3
                 ) {

  extern XSHARED float shared_data[];

  // blockDim.xy: [npixels_each_iteration, nfaces_each_iteration]
  int npixels_each_iteration = blockDim.x;
  int pixel_id_this_iteration = threadIdx.x;

  int nfaces_each_iteration = blockDim.y;
  int face_id_this_iteration = threadIdx.y;

  // niterations_each_block
  for (int pixel_iter_id = 0; pixel_iter_id < npixeliter_each_block;
       pixel_iter_id++) {

    /// GET PIXEL INDICES
    /// gridDim.x: [batch_size x (H x W / npixels_each_block)]
    /// ...         [npixeliter_each_block]
    /// blockDim.x: [npixels_each_iteration]
    int global_pixel_id = (blockIdx.x * npixeliter_each_block + pixel_iter_id) *
                              npixels_each_iteration +
                          pixel_id_this_iteration;
    if (global_pixel_id >= batch_size * H * W) { // not a valid pixel here
      continue;
    }
    int pixel_x = global_pixel_id % W;
    int pixel_y = (global_pixel_id / W) % H;
    int batch_id = (global_pixel_id / W / H) % batch_size;

    /// RESULTS ON THIS PIXEL
    float best_z_this_thread = -FLT_MAX;
    int best_fid_this_thread = -1;
    float best_bc_this_thread[3] = {-1.0f, -1.0f, -1.0f};

    for (int face_iter_id = 0; face_iter_id < nfaceiter_each_block;
         face_iter_id++) {

      /// GET FACE INDICES
      /// nfaces: nfaceiter_each_block x nfaces_each_iteration
      /// face_id: face_iter_id, face_id_this_iteration
      int face_id =
          face_iter_id * nfaces_each_iteration + face_id_this_iteration;
      if (face_id >= nfaces) {
        continue;
      }

      /// COMPUTE Z
      // get 3 corner point positions
      int32_t pids[3];
      float ppos[3][3];
      for (int k = 0; k < 3; k++) {
        pids[k] = faces_data[(batch_id * nfaces + face_id) * 3 + k];
        for (int j = 0; j < 3; j++) {
          ppos[k][j] = pts_data[(batch_id * npoints + pids[k]) * 3 + j];
        }
      }
      // compute barycentric coords
      float pixel_fx = (pixel_x + 0.5f) / W;
      pixel_fx = pixel_fx * 2 - 1; // [-1, 1]
      float pixel_fy = (pixel_y + 0.5f) / H;
      pixel_fy = pixel_fy * 2 - 1; // [-1, 1]
      float pixel_f[2] = {pixel_fx, pixel_fy};
      float bc[3];
      get_barycentric_coord(pixel_f, ppos[0], ppos[1], ppos[2], bc);

      // get z depth
      if (is_in_triangle(pixel_f, ppos[0], ppos[1], ppos[2])) {
        float z = 0;
        for (int k = 0; k < 3; k++) {
          z += bc[k] * ppos[k][2];
        }
        if (z >= best_z_this_thread) {
          best_z_this_thread = z;
          best_fid_this_thread = face_id;
          for (int k = 0; k < 3; k++) {
            best_bc_this_thread[k] = bc[k];
          }
        }
      }
    }

    // get the shared memory for storing z values on this pixel
    float *z_at_this_pixel =
        shared_data +
        2 * nfaces_each_iteration *
            pixel_id_this_iteration; // [nfaces_each_iteration]
    float *z_for_reduction =
        z_at_this_pixel + nfaces_each_iteration; // [nfaces_each_iteration]
    z_at_this_pixel[face_id_this_iteration] =
        z_for_reduction[face_id_this_iteration] = best_z_this_thread;
    __syncthreads();

    /// FIND THE FACE WITH MAXIMUM Z
    // find the max z and store it to z_for_reduction[0]
    for (unsigned int s = (nfaces_each_iteration + 1) / 2;; s = (s + 1) / 2) {
      if (face_id_this_iteration < s &&
          face_id_this_iteration + s < nfaces_each_iteration) {
        z_for_reduction[face_id_this_iteration] =
            max(z_for_reduction[face_id_this_iteration],
                z_for_reduction[face_id_this_iteration + s]);
      }
      __syncthreads();
      if (s == 1) {
        break;
      }
    }

    /// WRITE RESULTS
    out_z_data[global_pixel_id] = z_for_reduction[0];
    out_uvgrid_data[global_pixel_id * 2 + 0] = -1;
    out_uvgrid_data[global_pixel_id * 2 + 1] = -1;
    out_fids_data[global_pixel_id] = -1;
    __syncthreads();

    if (best_z_this_thread >= z_for_reduction[0]) { // this is the best thread
      out_fids_data[global_pixel_id] = best_fid_this_thread; // write face_id
      if (best_fid_this_thread != -1) {
        for (int k = 0; k < 3; k++) {
          out_bc_data[global_pixel_id * 3 + k] =
              best_bc_this_thread[k]; // write bc
        }

        float puvs[3][2];
        int best_global_face_id = batch_id * nfaces + best_fid_this_thread;
        for (int k = 0; k < 3; k++) {
          puvs[k][0] = uvs_data[(best_global_face_id * 3 + k) * 2 + 0];
          puvs[k][1] = uvs_data[(best_global_face_id * 3 + k) * 2 + 1];
        }
        float u = 0, v = 0;
        for (int k = 0; k < 3; k++) {
          u += best_bc_this_thread[k] * puvs[k][0];
          v += best_bc_this_thread[k] * puvs[k][1];
        }
        // write uvgrid
        out_uvgrid_data[global_pixel_id * 2 + 0] = u;
        out_uvgrid_data[global_pixel_id * 2 + 1] = v;
      }
    }
  }
}

template <>
void RasterizeOp<GPUDevice>::rasterize_impl(
    int batch_size, int npoints, int nfaces, const float *pts_data,
    const int32_t *faces_data, const float *uvs_data, int H, int W,
    float *out_uvgrid_data, float *out_z_data, int32_t *out_fids_data,
    float *out_bc_data) {

  int npixels = batch_size * H * W;

  int nblocks = min(kMaxGridNum, npixels);
  int npixels_each_block = (npixels + nblocks - 1) / nblocks;

  int nfaces_each_iteration = min(kMaxThreadsPerBlock, nfaces);
  int nfaceiter_each_block =
      (nfaces + nfaces_each_iteration - 1) / nfaces_each_iteration;

  int npixels_each_iteration =
      min(kMaxThreadsPerBlock / nfaces_each_iteration, npixels_each_block);
  int npixeliter_each_block =
      (npixels_each_block + npixels_each_iteration - 1) /
      npixels_each_iteration;

  dim3 grid_dim(nblocks);
  CHECK_LE(nblocks, kMaxGridNum);

  const unsigned shared_data_bytes =
      2 * nfaces_each_iteration * npixels_each_iteration * sizeof(float);
  dim3 block_dim(npixels_each_iteration, nfaces_each_iteration);
  CHECK_LE(npixels_each_iteration * nfaces_each_iteration, kMaxThreadsPerBlock);
  CHECK_LE(shared_data_bytes, 64 * 1024);

  XINVOKE_KERNEL(rasterize_kernel, grid_dim, block_dim, shared_data_bytes)
  (batch_size, npixeliter_each_block, nfaceiter_each_block, npoints, nfaces,
   pts_data, faces_data, uvs_data, H, W, out_uvgrid_data, out_z_data,
   out_fids_data, out_bc_data);
}

REGISTER_KERNEL_BUILDER(Name("Rasterize").Device(DEVICE_GPU),
                        RasterizeOp<GPUDevice>)

// a simple version of grad
struct rasterize_direct_grad_kernel {
  XDEVICE void
  operator()(int global_pixel_id, int batch_size, int nfaces, int npoints,
             int H, int W,
             const float *pts_data,         // batch_size x npoints x 3
             const int32_t *faces_data,     // batch_size x nfaces x 3
             const float *uvs_data,         // batch_size x nfaces x 3 x 2
             const int32_t *out_fids_data,  // batch_size x H x W
             const float *out_bc_data,      // batch_size x H x W x 3
             const float *grad_uvgrid_data, // batch_size x H x W x 2
             const float *grad_z_data,      // batch_size x H x W
             float *grad_pts                // batch_size x npoints x 3
             ) const {

    if (global_pixel_id >= batch_size * H * W) { // not a valid pixel here
      return;
    }
    int pixel_x = global_pixel_id % W;
    int pixel_y = (global_pixel_id / W) % H;
    int batch_id = (global_pixel_id / W / H) % batch_size;

    int face_id = out_fids_data[global_pixel_id];
    if (face_id == -1) {
      return;
    }

    // get 3 corner point positions
    int32_t pids[3];
    float ppos[3][3];
    for (int k = 0; k < 3; k++) {
      pids[k] = faces_data[(batch_id * nfaces + face_id) * 3 + k];
      for (int j = 0; j < 3; j++) {
        ppos[k][j] = pts_data[(batch_id * npoints + pids[k]) * 3 + j];
      }
    }

    float puvs[3][2];
    for (int k = 0; k < 3; k++) {
      puvs[k][0] = uvs_data[((batch_id * nfaces + face_id) * 3 + k) * 2 + 0];
      puvs[k][1] = uvs_data[((batch_id * nfaces + face_id) * 3 + k) * 2 + 1];
    }

    const float *bc = out_bc_data + global_pixel_id * 3;
    //  z = 0;
    //  for (int k = 0; k < 3; k++) {
    //    z += bc[k] * ppos[k][2];
    //  }
    const float grad_z = grad_z_data[global_pixel_id];
    //  float u = 0, v = 0;
    //  for (int k = 0; k < 3; k++) {
    //    u += bc[k] * puvs[k][0];
    //    v += bc[k] * puvs[k][1];
    //  }
    const float *grad_uvgrid = grad_uvgrid_data + global_pixel_id * 2;

    // grad_z -> grad_bc
    float grad_bc[3] = {0, 0, 0};
    for (int k = 0; k < 3; k++) {
      grad_bc[k] += grad_z * ppos[k][2];
    }
    // grad_uvgrid -> grad_bc
    for (int k = 0; k < 3; k++) {
      grad_bc[k] += grad_uvgrid[0] * puvs[k][0];
      grad_bc[k] += grad_uvgrid[1] * puvs[k][1];
    }

    float grad_points[3][3];
    for (int k = 0; k < 3; k++) {
      for (int j = 0; j < 3; j++) {
        grad_points[k][j] = 0;
      }
    }

    // grad_z -> grad_ppos[k][2]
    for (int k = 0; k < 3; k++) {
      grad_points[k][2] += grad_z * bc[k];
    }

    float pixel_fx = (pixel_x + 0.5f) / W;
    pixel_fx = pixel_fx * 2 - 1; // [-1, 1]
    float pixel_fy = (pixel_y + 0.5f) / H;
    pixel_fy = pixel_fy * 2 - 1; // [-1, 1]
    float pixel_f[2] = {pixel_fx, pixel_fy};
    // grad_bc -> grad_ppos[k][0, 1]
    add_barycentric_coord_grad(pixel_f, ppos[0], ppos[1], ppos[2], grad_bc,
                               grad_points[0], grad_points[1], grad_points[2]);

    // write results
    for (int k = 0; k < 3; k++) {
      int global_point_id = batch_id * npoints + pids[k];
      for (int j = 0; j < 3; j++) {
        atomicAdd(grad_pts + global_point_id * 3 + j, grad_points[k][j]);
      }
    }
  }
};

template <>
void RasterizeGradOp<GPUDevice>::rasterize_grad_impl(
    int batch_size, int nfaces, int npoints, int H, int W,
    const float *pts_data, const int32_t *faces_data, const float *uvs_data,
    const int32_t *out_fids_data, const float *out_bc_data,
    const float *grad_uvgrid_data, const float *grad_z_data,
    float *grad_pts_data) {
  Kernel<GPUDevice>::Launch(rasterize_direct_grad_kernel(), batch_size * H * W,
                            batch_size, nfaces, npoints, H, W, pts_data,
                            faces_data, uvs_data, out_fids_data, out_bc_data,
                            grad_uvgrid_data, grad_z_data, grad_pts_data);
}

REGISTER_KERNEL_BUILDER(Name("RasterizeGrad").Device(DEVICE_GPU),
                        RasterizeGradOp<GPUDevice>)

struct bilinear_sample_kernel {
  XINLINE void operator()(int idx, // BxHxW
                          int batch_size, int Ht, int Wt, int Dt, int H, int W,
                          const float *tex_data, // BxHtxWtxDt
                          const float *uvgrid_data, float *out_data) const {
    int batch_id = idx / W / H % batch_size;

    float u = uvgrid_data[idx * 2 + 0];
    float v = uvgrid_data[idx * 2 + 1];
    float uu_real = u * (Wt - 1) + 0.5;
    float vv_real = v * (Ht - 1) + 0.5;
    int uu = static_cast<int>(floorf(uu_real));
    int vv = static_cast<int>(floorf(vv_real));

    int uu2 = uu + 1;
    int vv2 = vv + 1;

    float uu_w = uu_real - uu;
    float vv_w = vv_real - vv;

    uu = min(Wt - 1, max(0, uu));
    vv = min(Ht - 1, max(0, vv));
    uu2 = min(Wt - 1, max(0, uu2));
    vv2 = min(Ht - 1, max(0, vv2));

    int tid_topleft = (batch_id * Ht + vv) * Wt + uu;
    int tid_bottomright = (batch_id * Ht + vv2) * Wt + uu2;
    int tid_topright = (batch_id * Ht + vv) * Wt + uu2;
    int tid_bottomleft = (batch_id * Ht + vv2) * Wt + uu;
    for (int k = 0; k < Dt; k++) {
      float color = tex_data[tid_topleft * Dt + k] * (1 - uu_w) * (1 - vv_w) +
                    tex_data[tid_topright * Dt + k] * uu_w * (1 - vv_w) +
                    tex_data[tid_bottomleft * Dt + k] * (1 - uu_w) * vv_w +
                    tex_data[tid_bottomright * Dt + k] * uu_w * vv_w;
      out_data[idx * Dt + k] = color;
    }
  }
};

template <>
void BilinearSampleOp<GPUDevice>::impl(int batch_size, int Ht, int Wt, int Dt,
                                       int H, int W, const float *tex_data,
                                       const float *uvgrid_data,
                                       float *out_data) {
  Kernel<GPUDevice>::Launch(bilinear_sample_kernel(), batch_size * H * W,
                            batch_size, Ht, Wt, Dt, H, W, tex_data, uvgrid_data,
                            out_data);
}

REGISTER_KERNEL_BUILDER(Name("BilinearSample").Device(DEVICE_GPU),
                        BilinearSampleOp<GPUDevice>)
